#include "hip/hip_runtime.h"
# include <eint_t.hpp>
# include <cstdio>
__global__ void cu_shift_left(mdl::uint_t *__xa_len, mdl::u8_t *__0pm, mdl::u8_t *__1pm) {
	mdl::uint_t point = (threadIdx.x + (blockIdx.x * *__xa_len)) * 3;
	__0pm[point] = __1pm[point + 3];
	__0pm[point + 1] = __1pm[point + 1 + 3];
	__0pm[point + 2] = __1pm[point + 2 + 3];
}

void shift_left(mdl::uint_t __xa_len, mdl::uint_t __ya_len, mdl::u8_t *__pm, mdl::uint_t __shift_amount) {
	bool static inited = false;
	mdl::u8_t static* _0pm = nullptr;
	mdl::u8_t static *_1pm = nullptr;
	mdl::uint_t static*xa_len;
	hipError_t any_err = hipSuccess;
	if (!inited) {
		if ((any_err = hipMalloc((void **)&xa_len, sizeof(mdl::uint_t))) != hipSuccess) {
			fprintf(stderr, "failed to alloc memory, errno: %d\n", any_err);
		}

		inited = true;
	}

	mdl::uint_t static _xa_len{};
	mdl::uint_t static _ya_len{};
	if (_xa_len != __xa_len || _ya_len != __ya_len) {
		if (_0pm != nullptr) hipFree(_0pm);
		if (_1pm != nullptr) hipFree(_1pm);

		hipMalloc((void **)&_0pm, (__xa_len * __ya_len) * 3);
		hipMalloc((void **)&_1pm, (__xa_len * __ya_len) * 3);
		if (_xa_len != __xa_len) {
			if ((any_err = hipMemcpy(xa_len, &__xa_len, sizeof(mdl::uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "failed to copy memory to device, errno: %d\n", any_err);
			}
		}

		_xa_len = __xa_len;
		_ya_len = __ya_len;
	}

	hipMemcpy(_0pm, __pm, (__xa_len * __ya_len) * 3, hipMemcpyHostToDevice);
	for (mdl::uint_t sa{}; sa != __shift_amount; sa ++) {
		hipMemcpy(_1pm, _0pm, (__xa_len * __ya_len) * 3, hipMemcpyDeviceToDevice);
		cu_shift_left<<<__ya_len, __xa_len - 1>>>(xa_len, _0pm, _1pm);
	}


	hipMemcpy(__pm, _0pm, (__xa_len * __ya_len) * 3, hipMemcpyDeviceToHost);
}

void cu_init() {

}

void cu_de_init() {

}


