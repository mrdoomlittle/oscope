#include "hip/hip_runtime.h"
# include "config.h"
# include <eint_t.hpp>
# include <cstdio>

__global__ void gpu_shift_left(mdl::uint_t *__xa_len, mdl::u8_t *__pm_frame, mdl::u8_t *__pmf_buff) {
	mdl::uint_t point = (threadIdx.x + (blockIdx.x * *__xa_len)) * 3;

	__pmf_buff[point] = __pm_frame[point + 3];
	__pmf_buff[point + 1] = __pm_frame[point + 1 + 3];
	__pmf_buff[point + 2] = __pm_frame[point + 2 + 3];
}

__global__ void gpu_draw_ib_buff(mdl::uint_t *__xa_len, mdl::u8_t *__pm_frame, mdl::u8_t *__ib_buff) {
	mdl::uint_t point = ((threadIdx.x + ((*__xa_len) - IB_BUFF_SIZE)) + (blockIdx.x * *__xa_len)) * 3;
	if (blockIdx.x < __ib_buff[threadIdx.x]) {
		__pm_frame[point] = 222;
		__pm_frame[point + 1] = 222;
		__pm_frame[point + 2] = 222;
	} else {
		__pm_frame[point] = 0;
		__pm_frame[point + 1] = 0;
		__pm_frame[point + 2] = 0;
	}
}

void build_frame(mdl::uint_t __xa_len, mdl::uint_t __ya_len, mdl::u8_t *__pm_frame, mdl::u8_t *__ib_buff) {
	bool static inited = false;
	mdl::uint_t static *xa_len;

	mdl::u8_t static *pm_frame = nullptr;
	mdl::u8_t static *pmf_buff = nullptr;
	mdl::u8_t static *ib_buff = nullptr;
	hipError_t any_err = hipSuccess;

	if (!inited) {
		if ((any_err = hipMalloc((void **)&xa_len, sizeof(mdl::uint_t))) != hipSuccess) {
			fprintf(stderr, "failed to alloc memory, errno: %d\n", any_err);
			return;
		}

		if ((any_err = hipMalloc((void **)&ib_buff, IB_BUFF_SIZE*sizeof(mdl::u8_t))) != hipSuccess) {
			fprintf(stderr, "failed to alloc memory, errno: %d\n", any_err);
			return;
		}

		inited = true;
	}

	mdl::uint_t static _xa_len{};
	mdl::uint_t static _ya_len{};

	if (_xa_len != __xa_len || _ya_len != __ya_len) {
		if (pm_frame != nullptr) hipFree(pm_frame);
		if (pmf_buff != nullptr) hipFree(pmf_buff);

		hipMalloc((void **)&pm_frame, (__xa_len * __ya_len) * 3);
		hipMalloc((void **)&pmf_buff, (__xa_len * __ya_len) * 3);

		if (_xa_len != __xa_len) {
			if ((any_err = hipMemcpy(xa_len, &__xa_len, sizeof(mdl::uint_t), hipMemcpyHostToDevice)) != hipSuccess) {
				fprintf(stderr, "failed to copy memory to device, errno: %d\n", any_err);
				return;
			}
		}

		hipMemset(pmf_buff, 0, (__xa_len * __ya_len) * 3);

		_xa_len = __xa_len;
		_ya_len = __ya_len;
	}

	hipMemcpy(pm_frame, __pm_frame, (__xa_len * __ya_len) * 3, hipMemcpyHostToDevice);
	hipMemcpy(ib_buff, __ib_buff, IB_BUFF_SIZE*sizeof(mdl::u8_t), hipMemcpyHostToDevice);

	gpu_draw_ib_buff<<<__ya_len, IB_BUFF_SIZE>>>(xa_len, pm_frame, ib_buff);

	for (mdl::uint_t sc{}; sc != IB_BUFF_SIZE; sc ++) {
		gpu_shift_left<<<__ya_len, __xa_len - 1>>>(xa_len, pm_frame, pmf_buff);
		hipMemcpy(pm_frame, pmf_buff, (__xa_len * __ya_len) * 3, hipMemcpyDeviceToDevice);
	}

	hipMemcpy(__pm_frame, pmf_buff, (__xa_len * __ya_len) * 3, hipMemcpyDeviceToHost);
}
